#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <time.h>
#include <sys/time.h>
#include <algorithm>

#define INF 1000000
#define ROOT 0

double t1, t2, COMM=0, COMP=0, MEM=0;
int n, m;	// Number of vertices, edges
int *Dist;
int *dev_dist;

double wallclock(void)
{	struct timeval tv;
	struct timezone tz;
	double t;

	gettimeofday(&tv, &tz);

	t = (double)tv.tv_sec*1000;
	t += ((double)tv.tv_usec)/1000.0;

	return t;
}// millisecond

void input(char *inFileName){	
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);

	//Dist = (int*)malloc(sizeof(int)*n*n);
	hipHostMalloc((void**) &Dist, sizeof(int)*n*n); //Pinned Memory
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			(i==j)?Dist[i*n+j]=0:Dist[i*n+j]=INF;

	int a, b, v;
	while (m--) {
		fscanf(infile, "%d %d %d", &a, &b, &v);
		Dist[(a-1)*n+(b-1)] = v;
	}
}

void output(char *outFileName){	
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			if (Dist[i*n+j] >= INF)	fprintf(outfile, "INF ");
			else					fprintf(outfile, "%d ", Dist[i*n+j]);
		}
		fprintf(outfile, "\n");
	}		
}

int ceil(int a, int b){	
	return (a + b -1)/b;
}

__global__ void cal(int n, int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int* dev_dist){	
	int block_end_x = block_start_x + block_height;
	int block_end_y = block_start_y + block_width;

	for (int b_i = block_start_x+blockIdx.x; b_i < block_end_x; b_i+=gridDim.x) {
		for (int b_j = block_start_y+blockIdx.y; b_j < block_end_y; b_j+=gridDim.y) {
			for (int k = Round * B; k < (Round +1) * B && k < n; k++) {
			
				int block_internal_start_x = b_i * B;
				int block_internal_end_x   = (b_i +1) * B;
				int block_internal_start_y = b_j * B; 
				int block_internal_end_y   = (b_j +1) * B;

				if (block_internal_end_x > n)	block_internal_end_x = n;
				if (block_internal_end_y > n)	block_internal_end_y = n;

				for (int i = block_internal_start_x+threadIdx.x; i < block_internal_end_x; i+=blockDim.x) {
					for (int j = block_internal_start_y+threadIdx.y; j < block_internal_end_y; j+=blockDim.y) {
						if (dev_dist[i*n+k] + dev_dist[k*n+j] < dev_dist[i*n+j])
							dev_dist[i*n+j] = dev_dist[i*n+k] + dev_dist[k*n+j];
					}
				}
				__syncthreads();
			}
		}
	}
}

void block_APSP(int B, int rank){
	hipSetDevice(1);
	hipMalloc((void**) &dev_dist, sizeof(int)*n*n);
	hipMemcpy(dev_dist, Dist,  sizeof(int)*n*n, hipMemcpyHostToDevice);

	float comp;
	double comm1, comm2, mem1, mem2;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	MPI_Request req;
	MPI_Status status;

	int round = ceil(n, B);
	dim3 block(round, round), thread(min(B,32), min(B,32));
	//dim3 block(10, 10), thread(10, 10);
	for (int r = 0; r < round; r++) {
		/* Phase 1*/
		hipEventRecord(start, 0);
		cal<<<block, thread>>>(n, B, r, r, r, 1, 1, dev_dist);
		if(rank==0){
			/*Phase 2*/
			cal<<<block, thread>>>(n, B, r, r, 0, r, 1, dev_dist);			//left
			cal<<<block, thread>>>(n, B, r, r, r+1, round-r-1, 1, dev_dist);//right
			cal<<<block, thread>>>(n, B, r, 0, r, 1, r, dev_dist);			//top
			/*Phase 3*/
			cal<<<block, thread>>>(n, B, r, 0, 0, r, r, dev_dist);						//left-top
			cal<<<block, thread>>>(n, B, r, 0, r+1, round-r-1, r, dev_dist);			//right-top
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&comp, start, stop);
			COMP+=comp;

			mem1 = wallclock();
			hipMemcpy(Dist, dev_dist, sizeof(int)*r*B*n, hipMemcpyDeviceToHost);
			mem2 = wallclock();
			MEM+=(mem2-mem1);

			comm1 = wallclock();
			MPI_Isend(Dist, r*B*n, MPI_INT, 1, 0, MPI_COMM_WORLD, &req);
			MPI_Recv(&Dist[r*B*n], (n-r*B)*n, MPI_INT, 1, 1, MPI_COMM_WORLD, &status);
			comm2 = wallclock();
			COMM+=(comm2-comm1);

			mem1 = wallclock();
			hipMemcpy(&dev_dist[r*B*n], &Dist[r*B*n],  sizeof(int)*(n-r*B)*n, hipMemcpyHostToDevice);
			mem2 = wallclock();
			MEM+=(mem2-mem1);

		}else{
			/*Phase 2*/
			cal<<<block, thread>>>(n, B, r, r, 0, r, 1, dev_dist);			//left
			cal<<<block, thread>>>(n, B, r, r, r+1, round-r-1, 1, dev_dist);//right
			cal<<<block, thread>>>(n, B, r, r+1, r, 1, round-r-1, dev_dist);//bottom
			/*Phase 3*/
			cal<<<block, thread>>>(n, B, r, r+1, 0, r, round-r-1, dev_dist);			//left-bottom
			cal<<<block, thread>>>(n, B, r, r+1, r+1, round-r-1, round-r-1, dev_dist);	//right-bottom
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&comp, start, stop);
			COMP+=comp;

			mem1 = wallclock();
			hipMemcpy(&Dist[r*B*n], &dev_dist[r*B*n], sizeof(int)*(n-r*B)*n, hipMemcpyDeviceToHost);
			mem2 = wallclock();
			MEM+=(mem2-mem1);

			comm1 = wallclock();
			MPI_Isend(&Dist[r*B*n], (n-r*B)*n, MPI_INT, 0, 1, MPI_COMM_WORLD, &req);
			MPI_Recv(Dist, r*B*n, MPI_INT, 0, 0, MPI_COMM_WORLD, &status);
			comm2 = wallclock();
			COMM+=(comm2-comm1);
			
			mem1 = wallclock();
			hipMemcpy(dev_dist, Dist,  sizeof(int)*r*B*n, hipMemcpyHostToDevice);
			mem2 = wallclock();
			MEM+=(mem2-mem1);
		}
	}
}

int main(int argc, char* argv[]){
	int rank, size;

	MPI_Init(&argc, &argv);
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	input(argv[1]);

	int B = 64;
	t1 = wallclock();
	block_APSP(B, rank);
	t2 = wallclock();
	/*printf("total[%d]  time %10.3lf\n", rank, t2-t1);
	printf("comp[%d]   time %10.3lf\n", rank, COMP);
	printf("comm[%d]   time %10.3lf\n", rank, COMM);
	printf("mem[%d]    time %10.3lf\n", rank, MEM);*/

	if(rank==ROOT) output(argv[2]);
	hipFree(dev_dist);
	MPI_Finalize();
	return 0;
}

